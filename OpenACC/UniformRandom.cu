//
// Created by jefferson on 21/09/16.
//

#include "UniformRandom.h"

#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>


UniformRandom::UniformRandom(double a, double b) :
        a(a), b(b)
{
}

thrust::device_vector<double> UniformRandom::generate(int n) {
    thrust::device_vector<double> result(static_cast<unsigned long>(n));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);

    thrust::transform(index_sequence_begin,
                      index_sequence_begin + n,
                      result.begin(),
                      prg_real(static_cast<unsigned int>(rand()), a, b));

    return result;
}
