#include "test.h"

using namespace std;
using ns = std::chrono::microseconds;
using get_time = std::chrono::steady_clock;

// Main routine that executes on the host
void test1() {
    Hipercube h(-5.12, 5.12, 10000);
    thrust::device_vector<double> a = h.getRandomIndividual();
    thrust::device_vector<double> b = h.getRandomIndividual();
    vector<thrust::device_vector<double>> inds;
    inds.push_back(a);
    inds.push_back(b);

    GaussianMutator gaussianMutator(0.0, 0.3, 0.1);
    LinearXOver linearXOver;
    auto start = get_time::now();
    for(int i = 0; i < 100; ++i) {
        h.repair(a);
    }
    auto end = get_time::now();
    auto diff = end - start;
    cout << std::chrono::duration_cast<ns>(diff).count() << endl;
    //test();
}

void test2() {
    size_t popSize[] = {100};
    for(int i = 0; i < 1; ++i) {
        std::cout << "population size: " << popSize[i] << std::endl;

        std::ofstream file;
        file.open(std::to_string(popSize[i]) + ".txt");
        size_t ITERS = 1000;
        size_t POP = popSize[i];
        size_t DIM = 1000;
        int sampling = 2;

        Hipercube space(-5.12, 5.12, static_cast<int>(DIM));

        Rastrigin optimizationFunction;
        Tournament selection(optimizationFunction, 4);

        std::shared_ptr<Operator<thrust::device_vector<double> > > lxo = std::make_shared<LinearXOver>(),
                    gm = std::make_shared<GaussianMutator>(0.0, 0.3, 0.1);


        std::vector< std::shared_ptr<Operator<thrust::device_vector<double> > > > opers(2);
        opers[0] = lxo;
        opers[1] = gm;

        AbstractHAEA<thrust::device_vector<double>> search(selection, opers, POP, ITERS);

        for(int k = 0; k < sampling; ++k) {
            std::cout << "iter: " << k << std::endl;
            auto start = get_time::now();
            search.solve(&space, &optimizationFunction);
            auto end = get_time::now();
            auto diff = end - start;
            cout << std::chrono::duration_cast<ns>(diff).count() << " ";
            file  << std::chrono::duration_cast<ns>(diff).count() << " ";
            /*thrust::host_vector<thrust::device_vector<double> > result = search.solve(&space, &optimizationFunction);
            double mean = 0.0;

            for(size_t i = 0; i < result.size(); ++i) {
                for(size_t j = 0; j < result[0].size(); ++j) {
                    cout << result[i][j] << " ";
                }
                mean += optimizationFunction.apply(result[i]);
            }

            mean /= result.size();
            rmean += mean;*/
        }
        file.close();
    }
}
