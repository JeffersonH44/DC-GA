#include "hip/hip_runtime.h"
//
// Created by jefferson on 21/09/16.
//

#include "UniformRandomInt.h"
#include <stdlib.h>



UniformRandomInt::UniformRandomInt(int a, int b) :
        a(a), b(b)
{
}

thrust::device_vector<double> UniformRandomInt::generate(int n) {
    thrust::device_vector<double> result(static_cast<unsigned long>(n));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);

    thrust::transform(index_sequence_begin,
                      index_sequence_begin + n,
                      result.begin(),
                      prg_int(static_cast<unsigned int>(rand()), a, b));

    return result;
}

void UniformRandomInt::test() {
    // Size of vectors
    int n = 10000;

    // Input vectors
    double *__restrict__ a;
    double *__restrict__ b;
    // Output vector
    double *__restrict__ c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);

    // Allocate memory for each vector
    a = (double*)malloc(bytes);
    b = (double*)malloc(bytes);
    c = (double*)malloc(bytes);

    // Initialize content of input vectors, vector a[i] = sin(i)^2 vector b[i] = cos(i)^2
    int i;
    for(i=0; i<n; i++) {
        a[i] = sin(i)*sin(i);
        b[i] = cos(i)*cos(i);
    }

    // sum component wise and save result into vector c
    #pragma acc kernels copyin(a[0:n],b[0:n]), copyout(c[0:n])
    for(i=0; i<n; i++) {
        c[i] = a[i] + b[i];
    }

    // Sum up vector c and print result divided by n, this should equal 1 within error
    double sum = 0.0;
    for(i=0; i<n; i++) {
        sum += c[i];
    }
    sum = sum/n;
    printf("final result: %f\n", sum);

    // Release memory
    free(a);
    free(b);
    free(c);
}
