#include "hip/hip_runtime.h"
//
// Created by jefferson on 21/09/16.
//

#include "Tournament.h"
#include "../../cpu/random/UniformRandomIntCPU.h"

Tournament::Tournament(OptimizationFunction<thrust::device_vector<double> > &function, int n) :
    eng(rd())
{
    this->function = &function;
    this->n = n;
}

size_t Tournament::chooseOne(thrust::host_vector< thrust::device_vector<double> > population) {
    UniformRandomIntCPU index(this->eng, 0, static_cast<int>(population.size()) - 1);

    double fitness = std::numeric_limits<double>::max();
    size_t bestIndex = 0;
    for(int i = 0; i < n; ++i) {
        size_t currIndex = static_cast<size_t>(index.generate());
        double currentFitness = function->apply(population[currIndex]);
        if(currentFitness < fitness) {
            fitness = currentFitness;
            bestIndex = currIndex;
        }
    }

    return bestIndex;
}