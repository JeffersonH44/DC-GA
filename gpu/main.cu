// Using CUDA device to calculate pi
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <fstream>
#include <chrono>

#include "functions/Rastrigin.h"
#include "random/UniformRandom.h"
#include "random/UniformRandomInt.h"
#include "random/GaussianRandom.h"
#include "operators/mutations/GaussianMutator.h"
#include "operators/xover/LinearXOver.h"
#include "ga/Hipercube.h"
#include "selection/Tournament.h"
#include "ga/HAEA/AbstractHAEA.h"

using namespace std;
using ns = std::chrono::microseconds;
using get_time = std::chrono::steady_clock;

void test();

// Main routine that executes on the host
int main() {
    Hipercube space(-5.12, 5.12, 100000);
    auto a = space.getRandomIndividual(); // zoloh auto lok
    Rastrigin opt;
    auto start = get_time::now();
    for(int i = 0 ; i < 10000; ++i) {
        opt.apply(a);
    }
    auto end = get_time::now();
    auto diff = end - start;
    cout << std::chrono::duration_cast<ns>(diff).count() << " ";
    return 0;
}

void test() {
    size_t popSize[] = {1000};
    for(int i = 0; i < 1; ++i) {
        std::cout << "population size: " << popSize[i] << std::endl;

        std::ofstream file;
        file.open(std::to_string(popSize[i]) + ".txt");
        size_t ITERS = 500;
        size_t POP = popSize[i];
        size_t DIM = 1000;
        int sampling = 2;

        Hipercube space(-5.12, 5.12, static_cast<int>(DIM));

        Rastrigin optimizationFunction;
        Tournament selection(optimizationFunction, 4);

        std::shared_ptr<Operator<thrust::device_vector<double> > > lxo = std::make_shared<LinearXOver>(), gm = std::make_shared<GaussianMutator>(0.0, 0.3, 0.1);


        std::vector< std::shared_ptr<Operator<thrust::device_vector<double> > > > opers(2);
        opers[0] = lxo;
        opers[1] = gm;

        AbstractHAEA<thrust::device_vector<double>> search(selection, opers, POP, ITERS);


        for(int k = 0; k < sampling; ++k) {
            std::cout << "iter: " << k << std::endl;
            auto start = get_time::now();
            search.solve(&space, &optimizationFunction);
            auto end = get_time::now();
            auto diff = end - start;
            file  << std::chrono::duration_cast<ns>(diff).count() << " ";
            /*thrust::host_vector<thrust::device_vector<double> > result = search.solve(&space, &optimizationFunction);
            double mean = 0.0;

            for(size_t i = 0; i < result.size(); ++i) {
                for(size_t j = 0; j < result[0].size(); ++j) {
                    cout << result[i][j] << " ";
                }
                mean += optimizationFunction.apply(result[i]);
            }

            mean /= result.size();
            rmean += mean;*/
        }
        file.close();
    }
}