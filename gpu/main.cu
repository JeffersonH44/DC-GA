// Using CUDA device to calculate pi
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
#include <fstream>
#include <chrono>

#include "functions/Rastrigin.h"
#include "random/UniformRandom.h"
#include "random/UniformRandomInt.h"
#include "random/GaussianRandom.h"
#include "operators/mutations/GaussianMutator.h"
#include "operators/xover/LinearXOver.h"
#include "ga/Hipercube.h"
#include "selection/Tournament.h"
#include "ga/HAEA/AbstractHAEA.h"

using namespace std;
using ns = std::chrono::microseconds;
using get_time = std::chrono::steady_clock;

void test();

// Main routine that executes on the host
int main() {
    Hipercube space(-5.12, 5.12, static_cast<int>(100000));

    cout << "here" << endl;
    std::vector< thrust::device_vector<double> > elem;
    cout << "here" << endl;
    for(int i = 0; i < 100; ++i) {
        cout << "here1" << endl;
        elem.push_back(space.getRandomIndividual());
        cout << "here2" << endl;
    }
    Rastrigin optimizationFunction;
    cout << "here" << endl;
    Tournament selection(optimizationFunction, 4);
    cout << "here" << endl;

    //std::shared_ptr<Operator<thrust::device_vector<double> > > lxo = std::make_shared<LinearXOver>(), gm = std::make_shared<GaussianMutator>(0.0, 0.3, 0.1);

    for(int i = 0; i < 1000; ++i) {
        cout << i << endl;
        selection.chooseOne(elem);
    }


    return 0;
}

void test() {
    size_t popSize[] = {50, 100, 200, 500, 1000};
    for(int i = 0; i < 5; ++i) {
        std::cout << "population size: " << popSize[i] << std::endl;
        for(size_t j = 1; j <= 8; j += 7) {

            std::ofstream file;
            file.open(std::to_string(popSize[i]) + "_" + std::to_string(j) + ".txt");

            std::cout << "threads: " << j << std::endl;
            size_t ITERS = 500;
            size_t POP = popSize[i];
            size_t DIM = 1000;
            int sampling = 35;

            Hipercube space(-5.12, 5.12, static_cast<int>(DIM));

            Rastrigin optimizationFunction;
            Tournament selection(optimizationFunction, 4);

            std::shared_ptr<Operator<thrust::device_vector<double> > > lxo = std::make_shared<LinearXOver>(), gm = std::make_shared<GaussianMutator>(0.0, 0.3, 0.1);


            thrust::host_vector< std::shared_ptr<Operator<thrust::device_vector<double> > > > opers(2);
            opers[0] = lxo;
            cout << "hello1" << endl;
            opers[1] = gm;
            cout << "hello1" << endl;

            AbstractHAEA<thrust::device_vector<double>> search(selection, opers, POP, ITERS);

            cout << "hello" << endl;

            for(int k = 0; k < sampling; ++k) {
                std::cout << "iter: " << k << std::endl;
                auto start = get_time::now();
                //search.solve(&space, &optimizationFunction);
                auto end = get_time::now();
                auto diff = end - start;
                file  << std::chrono::duration_cast<ns>(diff).count() << " ";
                /*thrust::host_vector<thrust::device_vector<double> > result = search.solve(&space, &optimizationFunction);
                double mean = 0.0;

                for(size_t i = 0; i < result.size(); ++i) {
                    for(size_t j = 0; j < result[0].size(); ++j) {
                        cout << result[i][j] << " ";
                    }
                    mean += optimizationFunction.apply(result[i]);
                }

                mean /= result.size();
                rmean += mean;*/
            }
            file.close();
        }
    }
}