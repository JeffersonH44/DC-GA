// Using CUDA device to calculate pi
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include "functions/Rastrigin.h"
#include "random/UniformRandom.h"
#include "random/UniformRandomInt.h"
#include "random/GaussianRandom.h"
#include "operators/mutations/GaussianMutator.h"

using namespace std;

// Main routine that executes on the host
int main() {
    thrust::host_vector< thrust::device_vector<double> > individuals, result;
    thrust::device_vector<double> ind(1000, 1.0);
    individuals.push_back(ind);

    GaussianMutator gm(0.0, 1.0, 10.0/50.0);
    for(int i = 0; i < 1000; ++i) {
        result = gm.apply(individuals);
        individuals = thrust::host_vector< thrust::device_vector<double> >(result.begin(), result.end());
    }


    for(size_t i = 0; i < ind.size(); ++i) {
        cout << individuals[0][i] << " ";
    }

    return 0;
}
