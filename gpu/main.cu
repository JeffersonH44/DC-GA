// Using CUDA device to calculate pi
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include "functions/Rastrigin.h"
#include "random/UniformRandom.h"
#include "random/UniformRandomInt.h"
#include "random/GaussianRandom.h"

using namespace std;

// Main routine that executes on the host
int main() {
    GaussianRandom r(5.0, 1.0);

    for(int j = 0 ; j < 2; ++j) {
        thrust::device_vector<double> res = r.generate(5 + j*5);
        for(size_t i = 0; i < res.size(); ++i) {
            cout << res[i] << endl;
        }
        cout << endl;
    }

    return 0;
}
