// Using CUDA device to calculate pi
#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>

#include "functions/Rastrigin.h"
#include "random/UniformRandom.h"
#include "random/UniformRandomInt.h"
#include "random/GaussianRandom.h"
#include "operators/mutations/GaussianMutator.h"
#include "operators/xover/LinearXOver.h"
#include "ga/Hipercube.h"

using namespace std;

// Main routine that executes on the host
int main() {
    Hipercube hipercube(-5.12, 5.12, 1000);

    thrust::host_vector< thrust::device_vector<double> > individuals;
    thrust::device_vector<double> ind = hipercube.getRandomIndividual(), ind1 = hipercube.getRandomIndividual();

    for(size_t i = 0; i < ind.size(); ++i) {
        cout << ind[i] << " " << ind1[i] << endl;
    }

    /*individuals.push_back(ind);
    individuals.push_back(ind1);

    LinearXOver lx;
    for(int i = 0; i < 2; ++i) {
        individuals = lx.apply(individuals);
    }


    for(size_t i = 0; i < ind.size(); ++i) {
        cout << individuals[0][i] << " " << individuals[1][i] << endl;
    }*/

    return 0;
}
