#include "hip/hip_runtime.h"
//
// Created by jefferson on 20/09/16.
//

#include "Rastrigin.h"

struct saxpy_rastrigin {
    __host__ __device__
    double operator() (const double& val) const {
        return val * val - 10.0 * cos(6.283185307179586 * val);
    }
};

double Rastrigin::apply(double val) {
    return val * val - 10.0 * cos(6.283185307179586 * val);
}

double Rastrigin::apply(thrust::device_vector<double> ind) {
    size_t len = ind.size();
    thrust::device_vector<double> partial_result(len);

    thrust::transform(ind.begin(), ind.end(), partial_result.begin(), saxpy_rastrigin());
    double fitness = thrust::reduce(partial_result.begin(), partial_result.end());
    /*for(size_t i = 0; i < len; ++i) {
        fitness += this->apply(ind[i]);
    }*/

    fitness = 10.0 * len + fitness;

    return std::abs(this->objective - fitness);
}


// TODO: be careful with the max double
Rastrigin::Rastrigin(bool minimize) {
    this->objective = minimize ? 0.0 : std::numeric_limits<double>::max();
}

Rastrigin::Rastrigin(double objective) {
    this->objective = objective;
}

Rastrigin::Rastrigin() :
        Rastrigin(true)
{
}