#include "hip/hip_runtime.h"
//
// Created by jefferson on 21/09/16.
//

#include "Hipercube.h"

Hipercube::Hipercube(double min, double max, int dimension) :
        ur(min, max),
        min(min),
        max(max),
        dimension(dimension)
{
}

thrust::device_vector<double> Hipercube::repair(thrust::device_vector<double> val){
    thrust::device_vector<double> ret(val.size());

    thrust::transform(ret.begin(), ret.end(), ret.begin(), rep(min, max));
    return ret;
}

thrust::device_vector<double> Hipercube::getRandomIndividual() {
    return (ur.generate(this->dimension));
}