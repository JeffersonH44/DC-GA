//
// Created by jefferson on 21/09/16.
//

#include "GaussianRandom.h"
#include <random>

GaussianRandom::GaussianRandom(double mean, double std) :
        mean(mean),
        std(std)
{
}

thrust::device_vector<double> GaussianRandom::generate(int n) {

    thrust::device_vector<double> result(static_cast<unsigned long>(n));
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    unsigned int seed = static_cast<unsigned int>(rand());

    thrust::transform(index_sequence_begin,
                      index_sequence_begin + n,
                      result.begin(),
                      prg_gaussian(seed, mean, std));

    return result;
}